#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <sstream>
#include <functional>
#include <direct.h>

#include "SimpleMath.h"
#include "cuda_smart_ptr.h"


__device__ float logistic(float signal) {
	return 1.0f - 2.0f * signal * signal;
}

/**
 * ������ ��������������� ����� �� ��������������� ������������,
 * ��������� � ������� ������� �� �����, ������ ����������� �������������.
 *
 * @param k
 * @param weightMatrix
 * @param neuronInput
 * @param neuronOutput
 *
 * ���������� ������ = ���-�� �������� � ��������� ����.
 */

__global__ void calcDynamicsOneThread(
	float *weightMatrix,
	float *neuronInput,
	float *output,
	int nNeurons
) {
	int neuronIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (neuronIdx >= nNeurons) {
		return;
	}

	float sum = 0.0f;
	float norm = 0.0f;
	
	for (int other = 0; other < nNeurons; other++) {
		// ��� ������ ������ ���� ������ ������ + L1 ���
		float prev = neuronInput[other];
		// ����� ��, ��������� ���������������� ������
		float w = weightMatrix[other * nNeurons + neuronIdx];
		norm += w;
		sum  += prev * w;
	}
	output[neuronIdx] = logistic((1.0f / norm) * sum);
}


// ������ ������� ������, � �� ���� �� ���������
#define SHARED_BLOCK_DIM_X 256

__global__ void calcDynamicsShared(
	float *weightMatrix,
	float *neuronInput,
	float *output,
	int nNeurons
) {
	int neuronIdx = blockIdx.x;

	if (neuronIdx >= nNeurons) {
		return;
	}

	float sum = 0.0f;
	float norm = 0.0f;

	for (int i = 0; i < nNeurons; i += blockDim.x) {
		int secondNeuron = i + threadIdx.x;
		if (secondNeuron < nNeurons) {
			// ��� ����� ������ ���� neuronInput ������������, ����������� �� ���������.
			float prev = neuronInput[secondNeuron];
			// ������ ���������������
			float w = weightMatrix[neuronIdx * nNeurons + secondNeuron];
			norm += w;
			sum  += prev * w;
		}
	}

	__shared__ float sums[SHARED_BLOCK_DIM_X];
	__shared__ float norms[SHARED_BLOCK_DIM_X];

	sums[threadIdx.x] = sum;
	norms[threadIdx.x] = norm;

	__syncthreads();

	// NOTE SHARED_BLOCK_DIM_X ������ ���� �������� 2.
	for (int stride = SHARED_BLOCK_DIM_X / 2; stride >= 1; stride = stride / 2) {
		if (threadIdx.x < stride && threadIdx.x + stride < SHARED_BLOCK_DIM_X) {
			sums[threadIdx.x]  +=  sums[threadIdx.x + stride];
			norms[threadIdx.x] += norms[threadIdx.x + stride];
		}
		__syncthreads();
	}

	__syncthreads();

	if (threadIdx.x == 0) {
		norm = norms[0];
		sum = sums[0];
		output[neuronIdx] = logistic((1.0f / norm) * sum);
	}
}


/*
������� ������������� (���������� �����).

������������� ����� ��������� �� 1 ����� ��������� ��� ����� ������ �������� �� �������, 
����� � �������� �� ������.
*/
__global__ void phaseSyncCheckInplace(int *currGT, int nSteps, int *hits, int nNeurons) {
	int neuronIdxFirst = threadIdx.x + blockIdx.x * blockDim.x;
	int neuronIdxSecond = threadIdx.y + blockIdx.y * blockDim.y;

	if (neuronIdxFirst >= nNeurons || neuronIdxSecond >= nNeurons) {
		return;
	}

	int count = 0;
	for (int step = 0; step < nSteps; step++) {
		int first = currGT[step * nNeurons + neuronIdxFirst];
		int second = currGT[step * nNeurons + neuronIdxSecond];
		if (first == second) {
			count++;
		}
	}
	hits[neuronIdxFirst * nNeurons + neuronIdxSecond] += count;
}


/**
���������� �������, ��� ����� ���������� ��� ���������� ������ ������� � ����� ��������������� ��������.
*/
__global__ void prepareToSynchronizationCheck(float *prevOutput, float *currOutput, int *gt, float *bufferedValues, int nNeurons) {
	int neuronIdx = threadIdx.x + blockIdx.x * blockDim.x;

	if (neuronIdx >= nNeurons) {
		return;
	}

	bool value = currOutput[neuronIdx] > prevOutput[neuronIdx];
	int result = 0;
	if (value) {
		result = 1;
	}
	gt[neuronIdx] = result;
	bufferedValues[neuronIdx] = currOutput[neuronIdx];
}


/**
����������� ��� �� ����� ����, ��� � � ������� �������������� - ��������� 
��������� ���������� �������, ��-�� ����� ���������� � ����.
*/
__global__ void fragmentaryAnalysis(float *currOutput, int nSteps, int *hits, int nNeurons, const float eps) {
	int neuronIdxFirst = threadIdx.x + blockIdx.x * blockDim.x;
	int neuronIdxSecond = threadIdx.y + blockIdx.y * blockDim.y;

	if (neuronIdxFirst >= nNeurons || neuronIdxSecond >= nNeurons) {
		return;
	}

	int count = 0;
	for (int step = 0; step < nSteps; step++) {
		float first = currOutput[step * nNeurons + neuronIdxFirst];
		float second = currOutput[step * nNeurons + neuronIdxSecond];
		float diff = fabsf(first - second);
		if (diff < eps) {
			count++;
		}
	}
	hits[neuronIdxFirst * nNeurons + neuronIdxSecond] += count;
}

__global__ void zeroInts(int *ar, int count) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= count) {
		return;
	}
	ar[idx] = 0;
}

inline int divRoundUp(int a, int b) {
	return (a - 1) / b + 1;
}

void randomSetHost(std::vector<float> &vals) {
	srand(23);
	for (int i = 0; i < static_cast<int>(vals.size()); i++) {
		vals[i] = (250 - rand() % 500) / 250.0f;
	}
}

void debugPrintArray(std::vector<float> &vals) {
	for (int j = 0; j < static_cast<int>(vals.size()); j++) {
		printf("%5.4f ", vals[j]);
	}
	printf("\r\n");
}

std::vector<int> processOscillatoryChaoticNetworkDynamics(
	int nNeurons,
	const std::vector<float> &weightMatrixHost,
	int startObservationTime,
	int nIterations,
	SyncType syncType,
	std::vector<float> &sheet,
	const float fragmentaryEPS,
	bool useSingleThreadPerNeuron
) {
	BEGIN_FUNCTION {
		check(nIterations > 0);
		check(nNeurons > 0);
		check(startObservationTime >= 0);

		DeviceScopedPtr1D<float> weightMatrixDevice(nNeurons * nNeurons);
		check(weightMatrixHost.size() == nNeurons * nNeurons);
		weightMatrixDevice.copyFromHost(&weightMatrixHost[0], weightMatrixHost.size());

		DeviceScopedPtr1D<float> input(nNeurons);
		DeviceScopedPtr1D<float> output(nNeurons);

		std::vector<float> stateHost(nNeurons);
		::randomSetHost(stateHost);

		input.copyFromHost(&stateHost[0], nNeurons);
		output.copyFromHost(&stateHost[0], nNeurons);
		
		float *currInputPtr = input.getDevPtr();
		float *currOutputPtr = output.getDevPtr();

		for (int i = 0; i < startObservationTime; i++) {
			if (useSingleThreadPerNeuron) {
				dim3 blockDim(256);
				dim3 gridDim(divRoundUp(nNeurons, blockDim.x));

				checkKernelRun((
					calcDynamicsOneThread<<<gridDim, blockDim>>>(
						weightMatrixDevice.getDevPtr(),
						currInputPtr,
						currOutputPtr,
						nNeurons
					)
				));
			} else {
				dim3 calcBlockDim(SHARED_BLOCK_DIM_X);
				dim3 calcGridDim(nNeurons);
				checkKernelRun((
					calcDynamicsShared<<<calcGridDim, calcBlockDim>>>(
						weightMatrixDevice.getDevPtr(),
						currInputPtr,
						currOutputPtr,
						nNeurons
					)
				));
			}
			std::swap(currInputPtr, currOutputPtr);
		}

		DeviceScopedPtr1D<int> hits(nNeurons * nNeurons);
		{
			dim3 blockDimFill(512);
			dim3 gridDimFill(divRoundUp(nNeurons * nNeurons, blockDimFill.x));
			checkKernelRun((
				zeroInts<<<gridDimFill, blockDimFill>>>(
					hits.getDevPtr(),
					nNeurons * nNeurons
				)
			));
		}

		DeviceScopedPtr1D<int> currentHits(nNeurons);
		std::vector<int> currentHitsHost(nNeurons);

		const int N_STEPS = 64;	
		DeviceScopedPtr1D<int> gt(nNeurons * N_STEPS);
		DeviceScopedPtr1D<float> bufferedValues(nNeurons * N_STEPS);

		sheet.resize(nIterations * nNeurons);

		int currentStep = 0;
		for (int i = 0; i < nIterations; i++) {
			// fragmentary synchronization if needed
			if (syncType == FRAGMENTARY && currentStep == N_STEPS)  {
				dim3 blockCheck(32, 8);
				dim3 gridCheck(divRoundUp(nNeurons, blockCheck.x), divRoundUp(nNeurons, blockCheck.y));

				checkKernelRun((
					fragmentaryAnalysis<<<gridCheck, blockCheck>>>(
						bufferedValues.getDevPtr(),
						N_STEPS,
						hits.getDevPtr(),
						nNeurons,
						fragmentaryEPS
					)
				));
				currentStep = 0;
			}

			// computing
			if (useSingleThreadPerNeuron) {
				dim3 calcBlockDim(256);
				dim3 calcGridDim(divRoundUp(nNeurons, calcBlockDim.x));
				checkKernelRun((
					calcDynamicsOneThread<<<calcGridDim, calcBlockDim>>>(
						weightMatrixDevice.getDevPtr(),
						currInputPtr,
						currOutputPtr,
						nNeurons
					)
				));
			} else {
				dim3 calcBlockDim(SHARED_BLOCK_DIM_X);
				dim3 calcGridDim(nNeurons);
				checkKernelRun((
					calcDynamicsShared<<<calcGridDim, calcBlockDim>>>(
						weightMatrixDevice.getDevPtr(),
						currInputPtr,
						currOutputPtr,
						nNeurons
					)
				));
			}
		
			{
				dim3 blockDim(128);
				dim3 gridDim(divRoundUp(nNeurons, blockDim.x));
				checkKernelRun((
					prepareToSynchronizationCheck<<<gridDim, blockDim>>>(
						currInputPtr,
						currOutputPtr,
						gt.getDevPtr() + nNeurons * currentStep,
						bufferedValues.getDevPtr() + nNeurons * currentStep,
						nNeurons
					)
				));
				currentStep++;
			}

			// phase synchronization if needed
			if (syncType == PHASE) {
				
				if (currentStep == N_STEPS) {
					dim3 blockCheck(32, 8);
					dim3 gridCheck(divRoundUp(nNeurons, blockCheck.x), divRoundUp(nNeurons, blockCheck.y));
					checkKernelRun((
						phaseSyncCheckInplace<<<gridCheck, blockCheck>>>(
							gt.getDevPtr(),
							N_STEPS,
							hits.getDevPtr(),
							nNeurons
						)
					));
					currentStep = 0;
				}
			}

			// copying neuron's outputs to host for sheets
			if (output.getDevPtr() == currOutputPtr) {
				output.copyToHost(&sheet[i * nNeurons], nNeurons);
			} else {
				input.copyToHost(&sheet[i * nNeurons], nNeurons);
			}
			// swapping pointers of input/output
			std::swap(currInputPtr, currOutputPtr);
		}

		// for phase synchronization if needed to procede remained
		if (currentStep != 0) {
			if (syncType == PHASE) {
				dim3 blockCheck(32, 8);
				dim3 gridCheck(divRoundUp(nNeurons, blockCheck.x), divRoundUp(nNeurons, blockCheck.y));
				checkKernelRun((
					phaseSyncCheckInplace<<<gridCheck, blockCheck>>>(
						gt.getDevPtr(),
						currentStep,
						hits.getDevPtr(),
						nNeurons
					)
				));
				currentStep = 0;
			} else if (syncType == FRAGMENTARY) {
				dim3 fragBlockDim(128);
				dim3 fragGridDim(divRoundUp(nNeurons, fragBlockDim.x));
				
				checkKernelRun((
					fragmentaryAnalysis<<<fragGridDim, fragBlockDim>>>(
						currInputPtr,
						currentStep,
						hits.getDevPtr(),
						nNeurons,
						fragmentaryEPS
					)
				));
			} else {
				throw std::logic_error("unknown sync type");
			}
		}
		std::vector<int> hitsHost(nNeurons * nNeurons);
		hits.copyToHost(&hitsHost[0], nNeurons * nNeurons);
		return hitsHost;
	} END_FUNCTION
}
