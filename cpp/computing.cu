#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <sstream>
#include <functional>
#include <direct.h>

#include "SimpleMath.h"
#include "cuda_smart_ptr.h"


__device__ float logistic(float signal) {
	return 1.0f - 2.0f * signal * signal;
}

/**
 * ������ ��������������� ����� �� ��������������� ������������,
 * ��������� � ������� ������� �� �����, ������ ����������� �������������.
 *
 * @param k
 * @param weightMatrix
 * @param neuronInput
 * @param neuronOutput
 *
 * ���������� ������ = ���-�� �������� � ��������� ����.
 */

__global__ void calcDynamics(
	float *weightMatrix,
	float *neuronInput,
	float *output,
	int nNeurons
) {
	int neuronIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (neuronIdx >= nNeurons) {
		return;
	}
	float sum = 0.0f;
	float norm = 0.0f;
	for (int i = 0; i < nNeurons; i++) {
		if (i != neuronIdx) {
			float w = weightMatrix[i * nNeurons + neuronIdx];
			norm += w;
			// ����� ����������������, ���� ��������� �� 1 neuron per thread, � ������.
			float prev = neuronInput[i];
			sum += logistic(prev) * w;
		}
	}
	output[neuronIdx] = (1.0f / norm) * sum;
}

/*
������� ������������� (���������� �����).

������ ������, ��������, ���������� �������� �� �����������. ��, �� ����, ���������� �����
��� ��� �� ��������� ������������ � ���-�� �������, ��� O(N^2).
*/

__global__ void phaseSyncCheckInplace(int *currGT, int *hits, int nNeurons) {
	int neuronIdxFirst = threadIdx.x + blockIdx.x * blockDim.x;
	int neuronIdxSecond = threadIdx.y + blockIdx.y * blockDim.y;

	if (neuronIdxFirst >= nNeurons || neuronIdxSecond >= nNeurons) {
		return;
	}

	// ������ ���� coalesced, �� ���� ����� ��������, �������� ILP.
	int first = currGT[neuronIdxFirst];
	int second = currGT[neuronIdxSecond];

	if (first == second) {
		hits[neuronIdxFirst * nNeurons + neuronIdxSecond]++;
	}
}


/**
���������� �������, �������� ����� ������.
*/
__global__ void phaseSyncCheck(float *prevOutput, float *currOutput, int *gt, int nNeurons) {
	int neuronIdx = threadIdx.x + blockIdx.x * blockDim.x;

	if (neuronIdx >= nNeurons) {
		return;
	}

	bool value = currOutput[neuronIdx] > prevOutput[neuronIdx];
	int result = 0;
	if (value) {
		result = 1;
	}
	gt[neuronIdx] = result;
}


/*
����� N ��������, ������� thread ���� 1 ������.
���� �� ���� ��������� ��������, ��������� ������� ��������
�������, ���� ����������� ������ �����������, �� ����������� �������
"�����" ������ ���� ��������.

� �� ������ �� �� �� ����� ������ �����������, �??
��� ���� �� O(N Log N) �������������
*/
__global__ void dynamicsAnalysis(
	float *currOutput,
	int *nHits,
	int *nCurrentStepHits,
	int nNeurons,
	const float eps
) {
	int neuronIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (neuronIdx >= nNeurons) {
		return;
	}

	int hitsCount = 0;
	float curr = currOutput[neuronIdx];
	for (int oppositeIdx = neuronIdx + 1; oppositeIdx < nNeurons; oppositeIdx++) {
		// ��������� � shared-������, ����� ����? �� ����� � L1-��� ��������� ���������
		float opp = currOutput[oppositeIdx];
		float diff = fabsf(opp - curr);
		// equals
		if (diff < eps) {
			// �������� ��������
			nHits[neuronIdx * nNeurons + oppositeIdx]++;
			hitsCount++;
		}
	}
	nCurrentStepHits[neuronIdx] = hitsCount;
}

__global__ void zeroInts(int *ar, int count) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= count) {
		return;
	}
	ar[idx] = 0;
}

inline int divRoundUp(int a, int b) {
	return (a - 1) / b + 1;
}

void randomSetHost(std::vector<float> &vals) {
	srand(23);
	for (int i = 0; i < static_cast<int>(vals.size()); i++) {
		vals[i] = (250 - rand() % 500) / 250.0f;
	}
}

void debugPrintArray(std::vector<float> &vals) {
	for (int j = 0; j < static_cast<int>(vals.size()); j++) {
		printf("%5.4f ", vals[j]);
	}
	printf("\r\n");
}

std::vector<int> processOscillatoryChaoticNetworkDynamics(
	int nNeurons,
	const std::vector<float> &weightMatrixHost,
	int startObservationTime,
	int nIterations,
	SyncType syncType,
	const float fragmentaryEPS
) {
	BEGIN_FUNCTION {
		check(nIterations > 0);
		check(nNeurons > 0);
		check(startObservationTime >= 0);

		DeviceScopedPtr1D<float> weightMatrixDevice(nNeurons * nNeurons);
		check(weightMatrixHost.size() == nNeurons * nNeurons);
		weightMatrixDevice.copyFromHost(&weightMatrixHost[0], weightMatrixHost.size());

		DeviceScopedPtr1D<float> input(nNeurons);
		DeviceScopedPtr1D<float> output(nNeurons);

		std::vector<float> stateHost(nNeurons);
		::randomSetHost(stateHost);
		printf("INITIAL\r\n");
		::debugPrintArray(stateHost);

		input.copyFromHost(&stateHost[0], nNeurons);
		output.copyFromHost(&stateHost[0], nNeurons);
		dim3 blockDim(256);
		dim3 gridDim(divRoundUp(nNeurons, blockDim.x));
		float *ptrEven = input.getDevPtr();
		float *ptrOdd = output.getDevPtr();
		
		for (int i = 0; i < startObservationTime; i++) {
			checkKernelRun((
				calcDynamics<<<gridDim, blockDim>>>(
					weightMatrixDevice.getDevPtr(),
					ptrEven,
					ptrOdd,
					nNeurons
				)
			));
		//	if (output.getDevPtr() == ptrOdd) {
		//		output.copyToHost(&stateHost[0], stateHost.size());
		//	} else {
		//		input.copyToHost(&stateHost[0], stateHost.size());
		//	}
		//	::debugPrintArray(stateHost);
			std::swap(ptrEven, ptrOdd);
		}

		DeviceScopedPtr1D<int> hits(nNeurons * nNeurons);
		{
			dim3 blockDimFill(512);
			dim3 gridDimFill(divRoundUp(nNeurons * nNeurons, blockDimFill.x));
			checkKernelRun((
				zeroInts<<<gridDimFill, blockDimFill>>>(
					hits.getDevPtr(),
					nNeurons * nNeurons
				)
			));
		}
		DeviceScopedPtr1D<int> currentHits(nNeurons);
		std::vector<int> currentHitsHost(nNeurons);
		DeviceScopedPtr1D<int> gt(nNeurons);

		for (int i = 0; i < nIterations; i++) {
			if (syncType == FRAGMENTARY) {
				checkKernelRun((
					dynamicsAnalysis<<<gridDim, blockDim>>>(
						ptrEven,
						hits.getDevPtr(),
						currentHits.getDevPtr(),
						nNeurons,
						fragmentaryEPS
					)
				));
			}

			checkKernelRun((
				calcDynamics<<<gridDim, blockDim>>>(
					weightMatrixDevice.getDevPtr(),
					ptrEven,
					ptrOdd,
					nNeurons
				)
			));

			if (syncType == PHASE) {
				checkKernelRun((
					phaseSyncCheck<<<gridDim, blockDim>>>(
						ptrEven,
						ptrOdd,
						gt.getDevPtr(),
						nNeurons
					)
				));
				dim3 blockCheck(32, 8);
				dim3 gridCheck(divRoundUp(nNeurons, blockCheck.x), divRoundUp(nNeurons, blockCheck.y));
				checkKernelRun((
					phaseSyncCheckInplace<<<gridCheck, blockCheck>>>(
						gt.getDevPtr(),
						hits.getDevPtr(),
						nNeurons
					)
				));
			}
		//	if (output.getDevPtr() == ptrOdd) {
		//		output.copyToHost(&stateHost[0], stateHost.size());
		//	} else {
		//		input.copyToHost(&stateHost[0], stateHost.size());
		//}
		//	::debugPrintArray(stateHost);
			std::swap(ptrEven, ptrOdd);
		}

		std::vector<int> hitsHost(nNeurons * nNeurons);
		hits.copyToHost(&hitsHost[0], nNeurons * nNeurons);
		return hitsHost;
	} END_FUNCTION
}

